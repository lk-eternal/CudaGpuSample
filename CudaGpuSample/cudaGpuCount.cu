#include "hip/hip_runtime.h"
﻿#include <sstream>                    // String to number conversion
#include <windows.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h> 
#include <time.h>

#define EPOCHFILETIME   (116444736000000000UL)

//GPUスレッドでの実行関数
__global__ void cudaAdd(double *array_a, double *array_b, double *array_c, int arraySize)
{
	//スレッドの番号を計算
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < arraySize) {
		//スレッド番号を配列のインデックスとして、
		//配列aと配列bのこのインデックスの値の加算して、配列cに入れる。
		array_c[tid] = array_a[tid] + array_b[tid];
	}
}

//現在の時刻を取得
int64_t getCurrentTime()
{
	FILETIME ft;
	LARGE_INTEGER li;
	int64_t tt = 0;
	GetSystemTimeAsFileTime(&ft);
	li.LowPart = ft.dwLowDateTime;
	li.HighPart = ft.dwHighDateTime;
	tt = (li.QuadPart - EPOCHFILETIME) / 10 / 10;
	return tt;
}

//CPUで計算
void runCPU(double *array_a, double *array_b, double *array_c, int arraySize)
{
	//ループで加算する
	for (int i = 0; i < arraySize; i++) {
		array_c[i] = array_a[i] + array_b[i];
	}
}

//GPUで計算
extern "C" void runGPU(double *array_a, double *array_b, double *array_g, int arraySize) {
	int size = arraySize * sizeof(double);

	//GPUメモリのアロケーション
	double *array_a_g;
	double *array_b_g;
	double *array_c_g;
	hipMalloc(&array_a_g, size);
	hipMalloc(&array_b_g, size);
	hipMalloc(&array_c_g, size);

	//データをCPUメモリからGPUメモリへコピー
	hipMemcpy(array_a_g, array_a, size, hipMemcpyHostToDevice);
	hipMemcpy(array_b_g, array_b, size, hipMemcpyHostToDevice);

	//グリッドとスレッドブロックを当てる
	dim3 blockSize(16);
	dim3 gridSize((arraySize + blockSize.x - 1) / blockSize.x);

	//GPU関数を呼び出す
	cudaAdd << <gridSize, blockSize >> >(array_a_g, array_b_g, array_c_g, arraySize);

	//GPU計算を待つ
	hipDeviceSynchronize();

	//データをGPUメモリからCPUメモリへコピー
	hipMemcpy(array_g, array_c_g, size, hipMemcpyDeviceToHost);

	//GPU領域解放
	hipFree(array_a_g);
	hipFree(array_b_g);
	hipFree(array_c_g);
}


int main1(int, char *argv[])
{
	//GPU初期化
	hipFree(nullptr);

	int N = 100000000;
	int size = N * sizeof(double);
	printf("arraySize:%d\n",N);

	//CPUメモリのアロケーション
	double *array_a = new double[N];
	double *array_b = new double[N];
	double *array_c = new double[N];
	double *array_g = new double[N];

	//ランダムデータを作成
	srand(time(NULL));
	for (int i = 0; i < N; i++) {
		array_a[i] = (i + 1) * 1.2 * rand() / RAND_MAX;
		array_b[i] = (i + 1) * 1.2 * rand() / RAND_MAX;
	}

	//CPUで計算
	int64_t csT = getCurrentTime();
	runCPU(array_a, array_b, array_c, N);
	csT = getCurrentTime() - csT;
	printf("CPUTime:%d\n",csT);

	//GPUで計算
	int64_t gsT = getCurrentTime();
	runGPU(array_a, array_b, array_g, N);
	gsT = getCurrentTime() - gsT;
	printf("GPUTime:%d\n", gsT);

	//結果をチェック
	for (int i = 0; i < N; i++) {
		if (array_c[i] != array_g[i]) {
			printf("has error!\n");
			printf("cpu[%d]:%f\n", i, array_c[i]);
			printf("gpu[%d]:%f\n", i, array_g[i]);
			break;
		}
	}

	//CPU領域解放
	free(array_a);
	free(array_b);
	free(array_c);
	free(array_g);

	system("pause");
	return 0;
}